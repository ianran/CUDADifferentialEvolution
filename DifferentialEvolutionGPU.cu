#include "hip/hip_runtime.h"
/* Copyright 2017 Ian Rankin
*
* Permission is hereby granted, free of charge, to any person obtaining a copy of this
* software and associated documentation files (the "Software"), to deal in the Software
* without restriction, including without limitation the rights to use, copy, modify, merge,
* publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons
* to whom the Software is furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or
* substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
* DEALINGS IN THE SOFTWARE.
*/

// DifferentialEvolutionGPU.cu
// This file holds the GPU kernel functions required to run differential evolution.
// The software in this files is based on the paper:
// Differential Evolution - A Simple and Efficient Heuristic for Global Optimization over Continous Spaces,
// Rainer Storn, Kenneth Price (1996)
//
// But is extended upon for use with GPU's for faster computation times.
// This has been done previously in the paper:
// Differential evolution algorithm on the GPU with C-CUDA
// Lucas de P. Veronese, Renato A. Krohling (2010)
// However this implementation is only vaguly based on their implementation.
// Translation: I saw that the paper existed, and figured that they probably
// implemented the code in a similar way to how I was going to implement it.
// Brief read-through seemed to be the same way.
//
// The paralization in this software is done by using multiple cuda threads for each
// agent in the algorithm. If using smaller population sizes, (4 - 31) this will probably
// not give significant if any performance gains. However large population sizes are more
// likly to give performance gains.
//
// HOW TO USE:
// To implement a new cost function write the cost function in DifferentialEvolutionGPU.cu with the header
// __device float fooCost(const float *vec, const void *args)
// @param vec - sample parameters for the cost function to give a score on.
// @param args - any set of arguements that can be passed at the minimization stage
// NOTE: args any memory given to the function must already be in device memory.
//
// Go to the header and add a specifier for your cost functiona and change the COST_SELECTOR
// to that specifier. (please increment from previous number)
//
// Once you have a cost function find the costFunc function, and add into
// preprocessor directives switch statement
//
// ...
// #elif COST_SELECTOR == YOUR_COST_FUNCTION_SPECIFIER
//      return yourCostFunctionName(vec, args);
// ...
//


#include <hiprand/hiprand_kernel.h>


#include <hip/hip_runtime.h>
// for random numbers in a kernel
#include "DifferentialEvolutionGPU.h"

// for FLT_MAX
#include <cfloat>

#include <iostream>

// for clock()
#include <ctime>
#include <cmath>

// basic function for exiting code on CUDA errors.
// Does no special error handling, just exits the program if it finds any errors and gives an error message.
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}


// -----------------IMPORTANT----------------
// costFunc - this function must implement whatever cost function
// is being minimized.
// Feel free to delete all code in here.
// This is a bit of a hack and not elegant at all. The issue is that
// CUDA doesn't support function passing device code between host
// software. There is a possibilty of using virtual functions, but
// was concerned that the polymorphic function have a lot of overhead
// So instead use this super ugly method for changing the cost function.
//
// @param vec - the vector to be evaulated.
// @param args - a set of user arguments.
__device__ float quadraticFunc(const float *vec, const void *args)
{
    float x = vec[0]-3;
    
    float y = vec[1];
    return (x*x) + (y*y);
}

__device__ float costWithArgs(const float *vec, const void *args)
{
    const struct data *a = (struct data *)args;
    
    float x = vec[0];
    float y = vec[1];
    
    return x*x + y*y + 9 - (6*x) + a->arr[1] + a->v;
}

__device__ float costFunctionWithManyLocalMinima(const float *vec, const void *args)
{
    float x = vec[0];
    float y = vec[1];
    return -(cos(x) + cos(y)) + 0.2*(x*x) + 0.2*(y*y);
}

__device__ float cost3D(const float *vec, const void *args)
{
    float x = vec[0] - 3;
    float y = vec[1] - 1;
    float z = vec[2] + 3;
    return (x*x*x*x)- (2*x*x*x) + (z*z*z*z) + (y*y*y);
}




// costFunc
// This is a selector of the functions.
// Although this code is great for usabilty, by using the preprocessor directives
// for selecting the cost function to use this gives no loss in performance
// wheras a switch statement or function pointer would require extra instructions.
// also function pointers in CUDA are complex to work with, and particulary with the
// architecture used where a standard C++ class is used to wrap the CUDA kernels and
// handle most of the memory mangement used.
__device__ float costFunc(const float *vec, const void *args) {
#if COST_SELECTOR == QUADRATIC_COST
    return quadraticFunc(vec, args);
#elif COST_SELECTOR == COST_WITH_ARGS
    return costWithArgs(vec, args);
#elif COST_SELECTOR == MANY_LOCAL_MINMA
    return costFunctionWithManyLocalMinima(vec, args);
#else
#error Bad cost_selector given to costFunc in DifferentialEvolution function: costFunc
#endif
}












void printCudaVector(float *d_vec, int size)
{
    float *h_vec = new float[size];
    gpuErrorCheck(hipMemcpy(h_vec, d_vec, sizeof(float) * size, hipMemcpyDeviceToHost));

    std::cout << "{";
    for (int i = 0; i < size; i++) {
        std::cout << h_vec[i] << ", ";
    }
    std::cout << "}" << std::endl;
    
    delete[] h_vec;
}

__global__ void generateRandomVectorAndInit(float *d_x, float *d_min, float *d_max,
            float *d_cost, void *costArgs, hiprandState_t *randStates,
            int popSize, int dim, unsigned long seed)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= popSize) return;
    
    hiprandState_t *state = &randStates[idx];
    hiprand_init(seed, idx,0,state);
    for (int i = 0; i < dim; i++) {
        d_x[(idx*dim) + i] = (hiprand_uniform(state) * (d_max[i] - d_min[i])) + d_min[i];
    }

    d_cost[idx] = costFunc(&d_x[idx*dim], costArgs);
}


// This function handles the entire differentialEvolution, and calls the needed kernel functions.
// @param d_target - a device array with the current agents parameters (requires array with size popSize*dim)
// @param d_trial - a device array with size popSize*dim (worthless outside of function)
// @param d_cost - a device array with the costs of the last generation afterwards size: popSize
// @param d_target2 - a device array with size popSize*dim (worthless outside of function)
// @param d_min - a list of the minimum values for the set of parameters (size = dim)
// @param d_max - a list of the maximum values for the set of parameters (size = dim)
// @param randStates - an array of random number generator states. Array created using createRandNumGen funtion
// @param dim - the number of dimensions the equation being minimized has.
// @param popSize - this the population size for DE, or otherwise the number of agents that DE will use. (see DE paper for more info)
// @param CR - Crossover Constant used by DE (see DE paper for more info)
// @param F - the scaling factor used by DE (see DE paper for more info)
// @param costArgs - this a set of any arguments needed to be passed to the cost function. (must be in device memory already)
__global__ void evolutionKernel(float *d_target,
                                float *d_trial,
                                float *d_cost,
                                float *d_target2,
                                float *d_min,
                                float *d_max,
                                hiprandState_t *randStates,
                                int dim,
                                int popSize,
                                int CR, // Must be given as value between [0,999]
                                float F,
                                void *costArgs)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= popSize) return; // stop executing this block if
                                // all populations have been used
    hiprandState_t *state = &randStates[idx];
    
    // TODO: Better way of generating unique random numbers?
    int a;
    int b;
    int c;
    int j;
    //////////////////// Random index mutation generation //////////////////
    // select a different random number then index
    do { a = hiprand(state) % popSize; } while (a == idx);
    do { b = hiprand(state) % popSize; } while (b == idx || b == a);
    do { c = hiprand(state) % popSize; } while (c == idx || c == a || c == b);
    j = hiprand(state) % dim;
    
    ///////////////////// MUTATION ////////////////
    for (int k = 1; k <= dim; k++) {
        if ((hiprand(state) % 1000) < CR || k==dim) {
            // trial vector param comes from vector plus weighted differential
            d_trial[(idx*dim)+j] = d_target[(a*dim)+j] + (F * (d_target[(b*dim)+j] - d_target[(c*dim)+j]));
        } else {
            d_trial[(idx*dim)+j] = d_target[(idx*dim)+j];
        } // end if else for creating trial vector
        j = (j+1) % dim;
    } // end for loop through parameters
    
    float score = costFunc(&d_trial[idx*dim], costArgs);
    if (score < d_cost[idx]) {
        // copy trial into new vector
        for (j = 0; j < dim; j++) {
            d_target2[(idx*dim) + j] = d_trial[(idx*dim) + j];
            //printf("idx = %d, d_target2[%d] = %f, score = %f\n", idx, (idx*dim)+j, d_trial[(idx*dim) + j], score);
        }
        d_cost[idx] = score;
    } else {
        // copy target to the second vector
        for (j = 0; j < dim; j++) {
            d_target2[(idx*dim) + j] = d_target[(idx*dim) + j];
            //printf("idx = %d, d_target2[%d] = %f, score = %f\n", idx, (idx*dim)+j, d_trial[(idx*dim) + j], score);
        }
    }
} // end differentialEvolution function.


// This is the HOST function that handles the entire Differential Evolution process.
// This function handles the entire differentialEvolution, and calls the needed kernel functions.
// @param d_target - a device array with the current agents parameters (requires array with size popSize*dim)
// @param d_trial - a device array with size popSize*dim (worthless outside of function)
// @param d_cost - a device array with the costs of the last generation afterwards size: popSize
// @param d_target2 - a device array with size popSize*dim (worthless outside of function)
// @param d_min - a list of the minimum values for the set of parameters (size = dim)
// @param d_max - a list of the maximum values for the set of parameters (size = dim)
// @param h_cost - this function once the function is completed will contain the costs of final generation.
// @param randStates - an array of random number generator states. Array created using createRandNumGen funtion
// @param dim - the number of dimensions the equation being minimized has.
// @param popSize - this the population size for DE, or otherwise the number of agents that DE will use. (see DE paper for more info)
// @param maxGenerations - the max number of generations DE will perform (see DE paper for more info)
// @param CR - Crossover Constant used by DE (see DE paper for more info)
// @param F - the scaling factor used by DE (see DE paper for more info)
// @param costArgs - this a set of any arguments needed to be passed to the cost function. (must be in device memory already)
// @param h_output - the host output vector of function
void differentialEvolution(float *d_target,
                           float *d_trial,
                           float *d_cost,
                           float *d_target2,
                           float *d_min,
                           float *d_max,
                           float *h_cost,
                           void *randStates,
                           int dim,
                           int popSize,
                           int maxGenerations,
                           int CR, // Must be given as value between [0,999]
                           float F,
                           void *costArgs,
                           float *h_output)
{
    hipError_t ret;
    int power32 = ceil(popSize / 32.0) * 32;
    //std::cout << "power32 = " << power32 << std::endl;
    
    //std::cout << "min bounds = ";
    //printCudaVector(d_min, dim);
    //std::cout << "max bounds = ";
    //printCudaVector(d_max, dim);
    
    //std::cout << "Random vector" << std::endl;
    //printCudaVector(d_target, popSize*dim);
    //std::cout << "About to create random vecto" << std::endl;
    
    // generate random vector
    generateRandomVectorAndInit<<<1, power32>>>(d_target, d_min, d_max, d_cost,
                    costArgs, (hiprandState_t *)randStates, popSize, dim, clock());
    gpuErrorCheck(hipPeekAtLastError());
    //udaMemcpy(d_target2, d_target, sizeof(float) * dim * popSize, hipMemcpyDeviceToDevice);
    
    //std::cout << "Generayed random vector" << std::endl;
    
    //printCudaVector(d_target, popSize*dim);
    //std::cout << "printing cost vector" << std::endl;
    //printCudaVector(d_cost, popSize);
    
    for (int i = 1; i <= maxGenerations; i++) {
        //std::cout << i << ": generation = \n";
        //printCudaVector(d_target, popSize * dim);
        //std::cout << "cost = ";
        //printCudaVector(d_cost, popSize);
        //std::cout << std::endl;
        
        // start kernel for this generation
        evolutionKernel<<<1, power32>>>(d_target, d_trial, d_cost, d_target2, d_min, d_max,
                (hiprandState_t *)randStates, dim, popSize, CR, F, costArgs);
        gpuErrorCheck(hipPeekAtLastError());
        
        // swap buffers, places newest data into d_target.
        float *tmp = d_target;
        d_target = d_target2;
        d_target2 = tmp;
    } // end for (generations)
    
    ret = hipDeviceSynchronize();
    gpuErrorCheck(ret);
    ret = hipMemcpy(h_cost, d_cost, popSize * sizeof(float), hipMemcpyDeviceToHost);
    gpuErrorCheck(ret);
    //std::cout << "h_cost = {";
    
    // find min of last evolutions
    int bestIdx = -1;
    float bestCost = FLT_MAX;
    for (int i = 0; i < popSize; i++) {
        float curCost = h_cost[i];
        //std::cout << curCost << ", ";
        if (curCost <= bestCost) {
            bestCost = curCost;
            bestIdx = i;
        }
    }
    //std::cout << "}" << std::endl;
    
    //std::cout << "\n\n agents = ";
    //printCudaVector(d_target, popSize*dim);
    
    //std::cout << "Best cost = " << bestCost << " bestIdx = " << bestIdx << std::endl;
    
    // output best minimization.
    ret = hipMemcpy(h_output, d_target+(bestIdx*dim), sizeof(float)*dim, hipMemcpyDeviceToHost);
    gpuErrorCheck(ret);
}

// allocate the memory needed for random number generators.
void *createRandNumGen(int size)
{
    void *x;
    gpuErrorCheck(hipMalloc(&x, sizeof(hiprandState_t)*size));
    return x;
}









