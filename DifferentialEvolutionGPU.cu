#include "hip/hip_runtime.h"
/* Copyright 2017 Ian Rankin
*
* Permission is hereby granted, free of charge, to any person obtaining a copy of this
* software and associated documentation files (the "Software"), to deal in the Software
* without restriction, including without limitation the rights to use, copy, modify, merge,
* publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons
* to whom the Software is furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or
* substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
* DEALINGS IN THE SOFTWARE.
*/


#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
// for random numbers in a kernel
#include "DifferentialEvolutionGPU.h"

// for FLT_MAX
#include <cfloat>

#include <iostream>

// for clock()
#include <ctime>
#include <cmath>



// -----------------IMPORTANT----------------
// costFunc - this function must implement whatever cost function
// is being minimized.
// Feel free to delete all code in here.
// This is a bit of a hack and not elegant at all. The issue is that
// CUDA doesn't support function passing device code between host
// software. There is a possibilty of using virtual functions, but
// was concerned that the polymorphic function have a lot of overhead
// So instead use this super ugly method for changing the cost function.
//
// @param vec - the vector to be evaulated.
// @param args - a set of user arguments.
__device__ float costFunc(const float *vec, const void *args)
{
    const struct data *a = (struct data *)args;
    float x = vec[0];
    float y = vec[1];
    //return (x*x*x*x)- (2*x*x*x)+25;
    //float z = (2*y)-2;
    //return (x*x*x*x)- (2*x*x*x) + (z*z*z*z) + (y*y*y);
    //return -46.78;
    //return -(cos(x) + cos(y)) + 0.2*(x*x) + 0.2*(y*y);
    return a->arr[2] + (x*x) + (y*y) + a->v;
}













void printCudaVector(float *d_vec, int size)
{
    float *h_vec = new float[size];
    hipMemcpy(h_vec, d_vec, sizeof(float) * size, hipMemcpyDeviceToHost);
    
    std::cout << "{";
    for (int i = 0; i < size; i++) {
        std::cout << h_vec[i] << ", ";
    }
    std::cout << "}" << std::endl;
    
    delete[] h_vec;
}

__global__ void generateRandomVectorAndInit(float *d_x, float *d_min, float *d_max,
            float *d_cost, CostFunc_t costFuncPassed, void *costArgs, hiprandState_t *randStates,
            int popSize, int dim, unsigned long seed)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= popSize) return;
    
    float test[1] = {4};
    
    hiprandState_t *state = &randStates[idx];
    hiprand_init(seed, idx,0,state);
    for (int i = 0; i < dim; i++) {
        d_x[(idx*dim) + i] = (hiprand_uniform(state) * (d_max[i] - d_min[i])) + d_min[i];
    }
    d_cost[idx] = costFunc(&d_x[idx*dim], costArgs);
    //d_cost[idx] = costFunc(test, costArgs);
}

__global__ void evolutionKernel(CostFunc_t costFuncPassed,
                                float *d_target,
                                float *d_trial,
                                float *d_cost,
                                float *d_target2,
                                hiprandState_t *randStates,
                                int dim,
                                int popSize,
                                int maxGenerations,
                                int CR, // Must be given as value between [0,999]
                                float F,
                                void *costArgs)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= popSize) return; // stop executing this block if
                                // all populations have been used
    hiprandState_t *state = &randStates[idx];
    
    // TODO: Better way of generating unique random numbers?
    int a;
    int b;
    int c;
    int j;
    //////////////////// Random index mutation generation //////////////////
    // select a different random number then index
    do { a = hiprand(state) % popSize; } while (a == idx);
    do { b = hiprand(state) % popSize; } while (b == idx || b == a);
    do { c = hiprand(state) % popSize; } while (c == idx || c == a || c == b);
    j = hiprand(state) % dim;
    
    ///////////////////// MUTATION ////////////////
    for (int k = 1; k <= dim; k++) {
        if ((hiprand(state) % 1000) < CR || k==dim) {
            // trial vector param comes from vector plus weighted differential
            d_trial[(idx*dim)+j] = d_target[(a*dim)+j] + (F * (d_target[(b*dim)+j] - d_target[(c*dim)+j]));
        } else {
            d_trial[(idx*dim)+j] = d_target[(idx*dim)+j];
        } // end if else for creating trial vector
        j = (j+1) % dim;
    } // end for loop through parameters
    
    float score = costFunc(&d_trial[idx*dim], costArgs);
    if (score < d_cost[idx]) {
        // copy trial into new vector
        for (j = 0; j < dim; j++) { d_target2[(idx*dim) + j] = d_trial[(idx*dim) + j]; }
        d_cost[idx] = score;
    }
} // end differentialEvolution function.

void differentialEvolution(CostFunc_t costFunc,
                           float *d_target,
                           float *d_trial,
                           float *d_cost,
                           float *d_target2,
                           float *d_min,
                           float *d_max,
                           float *h_cost,
                           void *randStates,
                           int dim,
                           int popSize,
                           int maxGenerations,
                           int CR, // Must be given as value between [0,999]
                           float F,
                           void *costArgs,
                           float *h_output)
{
    int power32 = ceil(popSize / 32.0) * 32;
    //std::cout << "power32 = " << power32 << std::endl;
    
    // generate random vector
    generateRandomVectorAndInit<<<1, power32>>>(d_target, d_min, d_max, d_cost,
                    costFunc, costArgs, (hiprandState_t *)randStates, popSize, dim, clock());
    
    hipMemcpy(d_target2, d_target, sizeof(float) * dim * popSize, hipMemcpyDeviceToDevice);
    
    //printCudaVector(d_target, popSize*dim);
    //printCudaVector(d_cost, popSize);
    
    for (int i = 1; i <= maxGenerations; i++) {
        //std::cout << i << ": generation = \n";
        //printCudaVector(d_target, popSize * dim);
        //std::cout << "cost = ";
        //printCudaVector(d_cost, popSize);
        //std::cout << std::endl;
        
        // start kernel for this generation
        evolutionKernel<<<1, power32>>>(costFunc, d_target, d_trial, d_cost, d_target2, (hiprandState_t *)randStates,
                                        dim, popSize, maxGenerations, CR, F, costArgs);
        
        // swap buffers, places newest data into d_target.
        float *tmp = d_target;
        d_target = d_target2;
        d_target2 = tmp;
    } // end for (generations)
    
    hipMemcpy(h_cost, d_cost, popSize * sizeof(float), hipMemcpyDeviceToHost);
    
    //std::cout << "h_cost = {";
    
    // find min of last evolutions
    int bestIdx = -1;
    float bestCost = FLT_MAX;
    for (int i = 0; i < popSize; i++) {
        float curCost = h_cost[i];
        //std::cout << curCost << ", ";
        if (curCost <= bestCost) {
            bestCost = curCost;
            bestIdx = i;
        }
    }
    //std::cout << "}" << std::endl;
    
    // output best minimization.
    hipMemcpy(h_output, d_target+bestIdx, sizeof(float)*dim, hipMemcpyDeviceToHost);
}

// allocate the memory needed for random number generators.
void *createRandNumGen(int size)
{
    void *x;
    hipMalloc(&x, sizeof(hiprandState_t)*size);
    return x;
}









